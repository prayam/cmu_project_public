#include "hip/hip_runtime.h"
#include "l2norm_helper.h"


template<typename T>
__global__ void sqrtKernel(const gint n, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void sqrtKernel(
    const gint n,
    const float* x,
    float* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = sqrtf(x[i]);
    }
}

template<>
__global__ void sqrtKernel(
    const gint n,
    const __half* x,
    __half* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
         y[i] = hsqrt(x[i]);
    }
}

template<typename T>
__global__ void rsqrtKernel(const gint n, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void rsqrtKernel(
    const gint n,
    const float* x,
    float* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = rsqrtf(x[i]);
    }
}

template<>
__global__ void rsqrtKernel(
    const gint n,
    const __half* x,
    __half* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
         y[i] = hrsqrt(x[i]);
    }
}

template<typename T>
__global__ void maxKernel(const gint n, const T eps, const T* x, T* y)
{
    printf("Unsupported type.");
}

template<>
__global__ void maxKernel(
    const gint n,
    const float eps,
    const float* x,
    float* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        y[i] = fmaxf(x[i], eps);
    }
}

template<>
__global__ void maxKernel(
    const gint n,
    const __half eps,
    const __half* x,
    __half* y)
{
    for (gint i = blockIdx.x * blockDim.x + threadIdx.x;
         i < n; i += gridDim.x * blockDim.x)
    {
        if (__hgt(x[i], eps))
        {
          y[i] = x[i];
        } else {
          y[i] = eps;
        }
    }
}

template <typename T>
gboolean executeInference(
    hipStream_t stream,
    gint op_type,
    T eps,
    gint batch_size,
    gint C,
    gint H,
    gint W,
    const T* input,
    T* output)
{
    const gint length = C * H * W;
    for (gint n = 0; n < batch_size; ++n)
    {
        switch(op_type)
        {
          case operation_t::OP_TYPE_MAX:
            maxKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, eps, input, output);
            break;
          case operation_t::OP_TYPE_RSQRT:
            rsqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          case operation_t::OP_TYPE_SQRT:
            sqrtKernel<<<(length + 511) / 512, 512, 0, stream>>>(length, input, output);
            break;
          default:
            return 1;
        }
        // Move cursors
        input += length;
        output += length;
    }
    return 0;
}

gint L2NormHelper::enqueue(
    gint batchSize,
    const void* const* inputs,
    void** outputs,
    void* workspace,
    hipStream_t stream)
{
    (void) workspace;

    switch(mDataType)
    {
      case DataType::kFLOAT:
        if (!executeInference(stream, op_type, eps, batchSize, C, H, W,
                              (const float*)inputs[0], (float*)outputs[0]))
          {
            return 1;
          }
        break;
      case DataType::kHALF:
        if (!executeInference(stream, op_type, (__half)eps, batchSize, C, H, W,
                              (const __half*)inputs[0], (__half*)outputs[0]))
          {
            return 1;
          }
        break;
      default:
        return 1;
    }
    return 0;
}
